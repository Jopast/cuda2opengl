#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "defines.h"

#define GVC_CLIP3(L, H, v)  min((H), max((v), (L)))

__global__ void YUV2RGBConver(gpel_t *pYdata, gpel_t *pUdata, gpel_t *pVdata, uchar3 *OutData, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height){
        int Y, U, V;
        int idx_y = (height - 1 - y) * width + x;
        int idx_uv = ((height >> 1) - 1 - (y >> 1)) * (width >> 1) + (x >> 1);
        int out_pos = y * width + x;
        Y = pYdata[idx_y];
        U = pUdata[idx_uv] - 128;
        V = pVdata[idx_uv] - 128;

        int R = Y + V + ((V * 103) >> 8);
        int G = Y - ((U * 88) >> 8) - ((V * 183) >> 8);
        int B = Y + U + ((U * 198) >> 8);

        OutData[out_pos].x = GVC_CLIP3(0, 255, R);//R
        OutData[out_pos].y = GVC_CLIP3(0, 255, G);//G
        OutData[out_pos].z = GVC_CLIP3(0, 255, B);//B
    }
}

extern "C"
int gvcd_yuv2rgb(gpel_t *pYdata, gpel_t *pUdata, gpel_t *pVdata, uchar3 *OutData,
    int width, int height)
{
    dim3 grids((width + 63) >> 6, (height + 15) >> 4);
    dim3 threads(64, 16);

    YUV2RGBConver << <grids, threads >> >(pYdata, pUdata, pVdata, OutData, width, height);

    return 0;
}