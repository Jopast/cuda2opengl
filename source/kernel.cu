#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "defines.h"

#define CAVS2_CLIP3(L, H, v)  min((H), max((v), (L)))

__global__ void YUV2RGBConver(gpel_t *pYdata, gpel_t *pUdata, gpel_t *pVdata, uchar3 *OutData, int width, int height, int stride_y, int stride_uv)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height){
        int Y, U, V;
        int idx_y = (height - 1 - y) * stride_y + x;
        int idx_uv = ((height >> 1) - 1 - (y >> 1)) * stride_uv + (x >> 1);
        int out_pos = y * width + x;
        Y = pYdata[idx_y];
        U = pUdata[idx_uv] - 128;
        V = pVdata[idx_uv] - 128;

        int R = Y + V + ((V * 103) >> 8);
        int G = Y - ((U * 88) >> 8) - ((V * 183) >> 8);
        int B = Y + U + ((U * 198) >> 8);

        OutData[out_pos].x = CAVS2_CLIP3(0, 255, R);//R
        OutData[out_pos].y = CAVS2_CLIP3(0, 255, G);//G
        OutData[out_pos].z = CAVS2_CLIP3(0, 255, B);//B
    }
}

extern "C"
int gvcd_yuv2rgb(gpel_t *pYdata, gpel_t *pUdata, gpel_t *pVdata, uchar3 *OutData,
                 int width, int height, int stride_y, int stride_uv)
{
    dim3 grids((width + 15) >> 4, (height + 15) >> 4);
    dim3 threads(16, 16);

    YUV2RGBConver << <grids, threads >> >(pYdata, pUdata, pVdata, OutData, width, height, stride_y, stride_uv);

    return 0;
}